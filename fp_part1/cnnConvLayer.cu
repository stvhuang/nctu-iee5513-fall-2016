#include "hip/hip_runtime.h"
// This program executes a typical convolutional layer in regular CNNs
#include <iostream>
#include "cnnConvLayer.h"

using namespace std;

static const int FMAREA(FMSIZE * FMSIZE);
static const int FILTAREA(FILTSIZE * FILTSIZE);
static const int FILTVOL(FMDEPTH * FILTAREA);
static const int OUTSIZE(FMSIZE / 2);
static const int OUTAREA(OUTSIZE * OUTSIZE);
static const int DOUBLE_FMDEPTH(2 * FMDEPTH);
//static const int outArea = FMSIZE / 2 * FMSIZE / 2;

// This is the CPU version, please don't modify it
void convLayerCPU()
{
    const int filtVol = FMDEPTH * FILTSIZE * FILTSIZE;
    const int filtArea = FILTSIZE * FILTSIZE;
    const int fmArea = FMSIZE * FMSIZE;
    const int outArea = FMSIZE / 2 * FMSIZE / 2;

    int fn(0), fmy(0), fmx(0), sum(0), sli(0), y(0), x(0), ifmy(0), ifmx(0), filtIdx(0), inNeuIdx(0), outNeuIdx(0), max(0), tmpVal(0), ofmy(0), ofmx(0), outIdx(0);

    // Convolution
    for (fn = 0; fn < FILTNUM; ++fn)
    {
        for (fmy = 0; fmy < FMSIZE; fmy += STRIDE)
        {
            for (fmx = 0; fmx < FMSIZE; fmx += STRIDE)
            {
                sum = 0;
                for (sli = 0; sli < FMDEPTH; ++sli)
                {
                    for (y = 0; y < FILTSIZE; ++y)
                    {
                        for (x = 0; x < FILTSIZE; ++x)
                        {
                            ifmy = fmy - FILTSIZE / 2 + y;
                            ifmx = fmx - FILTSIZE / 2 + x;
                            filtIdx = fn * filtVol + sli * filtArea + y * FILTSIZE + x;
                            inNeuIdx = sli * fmArea + ifmy * FMSIZE + ifmx;
                            if (ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)
                            {
                                sum += filt[filtIdx] * inNeu[inNeuIdx];
                            }
                        }
                    }
                }
                // Activation - ReLU
                outNeuIdx = fn * fmArea + fmy * FMSIZE + fmx;
                if (sum <= 0)
                {
                    outNeu[outNeuIdx] = 0;
                }
                else
                {
                    outNeu[outNeuIdx] = sum;
                }
            }
        }
    }

    // Max Pooling with Window Size 2x2
    for (sli = 0; sli < FILTNUM; ++sli)
    {
        for (fmy = 0; fmy < FMSIZE / 2 ; ++fmy)
        {
            for (fmx = 0; fmx < FMSIZE / 2 ; ++fmx)
            {
                outNeuIdx = sli * fmArea + fmy * 2 * FMSIZE + fmx * 2;
                max = outNeu[outNeuIdx];
                for (y = 0; y < 2; ++y)
                {
                    for (x = 0; x < 2; ++x)
                    {
                        ofmy = fmy * 2 + y;
                        ofmx = fmx * 2 + x;
                        outNeuIdx = sli * fmArea + ofmy * FMSIZE + ofmx;
                        tmpVal = outNeu[outNeuIdx];
                        if (tmpVal > max)
                        {
                            max = tmpVal;
                        }
                    }
                }
                outIdx = sli * outArea + fmy * FMSIZE / 2 + fmx;
                outCPU[outIdx] = max;
            }
        }
    }
}

/*** Implement your CUDA Kernel here ***/
__global__
void convLayerGPU(short * inNeu, short * filt, int * outNeu, int * out)
{
    __shared__ short filter[FILTVOL];
    __shared__ int tmp[DOUBLE_FMDEPTH];

    int i, x, y, kx, ky, inx, iny, outx, outy, sum, max;
    int upLow(threadIdx.x % 2), newIdx(threadIdx.x / 2);

    if (upLow == 0)
    {
        for (i = 0; i < FILTAREA; i++)
        {
            filter[FILTAREA * newIdx + i] = filt[FILTVOL * blockIdx.x + FILTAREA * newIdx + i];
        }
    }

    __syncthreads();

    //if (threadIdx.x < 512)
    int y_start = upLow * 16; // 0 and 16
    int y_end = y_start + 16; // 16 and 32

    for (x = 0; x < FMSIZE; ++x)
    {
        for (y = y_start; y < y_end; ++y)
        {
            tmp[threadIdx.x] = 0;
            for (kx = 0; kx < FILTSIZE; ++kx)
            {
                for (ky = 0; ky < FILTSIZE; ++ky)
                {
                    inx = x - FILTSIZE / 2 + kx;
                    iny = y - FILTSIZE / 2 + ky;
                    if (inx >= 0 && inx < FMSIZE && iny >= 0 && iny < FMSIZE)
                    {
                        tmp[threadIdx.x] += inNeu[newIdx * FMAREA + iny * FMSIZE + inx] \
                            * filter[newIdx * FILTAREA + ky * FILTSIZE + kx];
                    }
                }
            }

            __syncthreads();

            sum = 0;
            if (threadIdx.x == 0 || threadIdx.x == 2)
            {
                for (i = newIdx; i < DOUBLE_FMDEPTH; i += 2)
                {
                    sum += tmp[i];
                }

                if (sum > 0)
                {
                    outNeu[blockIdx.x * FMAREA + (y + (16 * newIdx)) * FMSIZE + x] = sum;
                }
                else
                {
                    outNeu[blockIdx.x * FMAREA + (y + (16 * newIdx)) * FMSIZE + x] = 0;
                }
            }

            __syncthreads();
        }
    }

    __syncthreads();

    // Max Pooling

    max = 0;
    if (threadIdx.x < OUTAREA)
    {
        outy = threadIdx.x % OUTSIZE;
        outx = threadIdx.x / OUTSIZE;
        for (int i = 0; i < 2; ++i)
        {
            for (int j = 0; j < 2; ++j)
            {
                if (max < outNeu[blockIdx.x * FMAREA + (outy * 2 + i) * FMSIZE + (outx * 2 + j)])
                {
                    max = outNeu[blockIdx.x * FMAREA + (outy * 2 + i) * FMSIZE + (outx * 2 + j)];
                }
            }
        }
        out[blockIdx.x * OUTAREA + outy * OUTSIZE + outx] = max;
    }
}
/*** Implement your CUDA Kernel here ***/

int main()
{
  int convLayerCPUExecTime, convLayerGPUExecTime;
    init(); // Initialize the data on host memory

  timespec time_begin, time_end;

  clock_gettime(CLOCK_REALTIME, &time_begin);
    convLayerCPU();
  clock_gettime(CLOCK_REALTIME, &time_end);
  convLayerCPUExecTime = timespec_diff_us(time_begin, time_end);
  cout << "CPU time for executing a typical convolutional layer = " <<  convLayerCPUExecTime / 1000 << "ms" << endl;

    // declare device pointer
    short * devInputNeuron;
    short * devInputFilter;
    int * devOutputNeuron;
    int * devOutput;

    // compute the size for allocating memory on device
    const int inputNeuronSize = sizeof(short) * FMSIZE * FMSIZE * FMDEPTH;
    const int filtersSize = sizeof(short) * FILTNUM * FILTSIZE * FILTSIZE * FMDEPTH;
    const int outputNeuronSize = sizeof(int) * FMSIZE * FMSIZE * FMDEPTH;
    const int outputSize = sizeof(int) * FMSIZE / 2 * FMSIZE / 2 * FMDEPTH;

    // allocate memory on device
    hipMalloc(&devInputNeuron, inputNeuronSize);
    hipMalloc(&devInputFilter, filtersSize);
    hipMalloc(&devOutputNeuron, outputNeuronSize);
    hipMalloc(&devOutput, outputSize);

    // copy data from host to deivce
    hipMemcpy(devInputNeuron, inNeu, inputNeuronSize, hipMemcpyHostToDevice);
    hipMemcpy(devInputFilter, filt, filtersSize, hipMemcpyHostToDevice);

    /*** Lunch your CUDA Kernel here ***/
  clock_gettime(CLOCK_REALTIME, &time_begin);
    convLayerGPU<<<512, 1024>>>(devInputNeuron, devInputFilter, devOutputNeuron, devOutput); // Lunch the kernel
    hipDeviceSynchronize(); // Do synchronization before clock_gettime()
  clock_gettime(CLOCK_REALTIME, &time_end);
    /*** Lunch your CUDA Kernel here ***/

    // copy data from device back to host
    hipMemcpy(outGPU, devOutput, outputSize, hipMemcpyDeviceToHost);

    // free the allocated memory on device
    hipFree(&devInputNeuron);
    hipFree(&devOutputNeuron);
    hipFree(&devOutput);
    hipFree(&devInputFilter);

  convLayerGPUExecTime = timespec_diff_us(time_begin, time_end);
  cout << "GPU time for executing a typical convolutional layer = " << convLayerGPUExecTime / 1000 << "ms" << endl;

    if (checker())
    {
        cout << "Congratulations! You pass the check." << endl;
        cout << "Speedup: " << (float)convLayerCPUExecTime / convLayerGPUExecTime << endl;
    }
    else
    {
        cout << "Sorry! Your result is wrong." << endl;
    }

    ending();

    return 0;
}
