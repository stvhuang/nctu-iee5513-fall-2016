#include "hip/hip_runtime.h"
// This program executes a typical convolutional layer in regular CNNs
#include <iostream>
#include "cnnConvLayer.h"

using namespace std;

// This is the CPU version, please don't modify it
void convLayerCPU()
{
    const int filtVol = FMDEPTH * FILTSIZE * FILTSIZE;
    const int filtArea = FILTSIZE * FILTSIZE;
    const int fmArea = FMSIZE * FMSIZE;
    const int outArea = FMSIZE / 2 * FMSIZE / 2;

    int fn(0), fmy(0), fmx(0), sum(0), sli(0), y(0), x(0), ifmy(0), ifmx(0), filtIdx(0), inNeuIdx(0), outNeuIdx(0), max(0), tmpVal(0), ofmy(0), ofmx(0), outIdx(0);

    // Convolution
    for (fn = 0; fn < FILTNUM; ++fn)
    {
        for (fmy = 0; fmy < FMSIZE; fmy += STRIDE)
        {
            for (fmx = 0; fmx < FMSIZE; fmx += STRIDE)
            {
                sum = 0;
                for (sli = 0; sli < FMDEPTH; ++sli)
                {
                    for (y = 0; y < FILTSIZE; ++y)
                    {
                        for (x = 0; x < FILTSIZE; ++x)
                        {
                            ifmy = fmy - FILTSIZE / 2 + y;
                            ifmx = fmx - FILTSIZE / 2 + x;
                            filtIdx = fn * filtVol + sli * filtArea + y * FILTSIZE + x;
                            inNeuIdx = sli * fmArea + ifmy * FMSIZE + ifmx;
                            if (ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)
                            {
                                sum += filt[filtIdx] * inNeu[inNeuIdx];
                            }
                        }
                    }
                }
                // Activation - ReLU
                outNeuIdx = fn * fmArea + fmy * FMSIZE + fmx;
                if (sum <= 0)
                {
                    outNeu[outNeuIdx] = 0;
                }
                else
                {
                    outNeu[outNeuIdx] = sum;
                }
            }
        }
    }

    // Max Pooling with Window Size 2x2
    for (sli = 0; sli < FILTNUM; ++sli)
    {
        for (fmy = 0; fmy < FMSIZE / 2 ; ++fmy)
        {
            for (fmx = 0; fmx < FMSIZE / 2 ; ++fmx)
            {
                outNeuIdx = sli * fmArea + fmy * 2 * FMSIZE + fmx * 2;
                max = outNeu[outNeuIdx];
                for (y = 0; y < 2; ++y)
                {
                    for (x = 0; x < 2; ++x)
                    {
                        ofmy = fmy * 2 + y;
                        ofmx = fmx * 2 + x;
                        outNeuIdx = sli * fmArea + ofmy * FMSIZE + ofmx;
                        tmpVal = outNeu[outNeuIdx];
                        if (tmpVal > max)
                        {
                            max = tmpVal;
                        }
                    }
                }
                outIdx = sli * outArea + fmy * FMSIZE / 2 + fmx;
                outCPU[outIdx] = max;
            }
        }
    }
}

/*** Implement your CUDA Kernel here ***/
__global__
void convLayerGPU(short * devInputNeuron, short * devInputFilter, int * devOutputNeuron, int * devOutput)
{
    const int filtVol = FMDEPTH * FILTSIZE * FILTSIZE;
    const int filtArea = FILTSIZE * FILTSIZE;
    const int fmArea = FMSIZE * FMSIZE;
    const int outArea = FMSIZE / 2 * FMSIZE / 2;

    int fn(blockIdx.x), fmy(0), fmx(0), sum(0), sli(0), y(0), x(0), ifmy(0), ifmx(0), filtIdx(0), inNeuIdx(0), outNeuIdx(0);

    for (fmy = 0; fmy < FMSIZE; fmy += STRIDE)
    {
        for (fmx = 0; fmx < FMSIZE; fmx += STRIDE)
        {
            sum = 0;
            for (sli = 0; sli < FMDEPTH; ++sli)
            {
                for (y = 0; y < FILTSIZE; ++y)
                {
                    for (x = 0; x < FILTSIZE; ++x)
                    {
                        ifmy = fmy - FILTSIZE / 2 + y;
                        ifmx = fmx - FILTSIZE / 2 + x;
                        filtIdx = fn * filtVol + sli * filtArea + y * FILTSIZE + x;
                        inNeuIdx = sli * fmArea + ifmy * FMSIZE + ifmx;
                        if (ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)
                        {
                            sum += devInputFilter[filtIdx] * devInputNeuron[inNeuIdx];
                        }
                    }
                }
            }
            // Activation - ReLU
            outNeuIdx = fn * fmArea + fmy * FMSIZE + fmx;
            if (sum <= 0)
            {
                devOutputNeuron[outNeuIdx] = 0;
            }
            else
            {
                devOutputNeuron[outNeuIdx] = sum;
            }
        }
    }

    // Max Pooling with Window Size 2x2
    int max(0), tmpVal(0), ofmy(0), ofmx(0), outIdx(0);
    for (sli = 0; sli < FILTNUM; ++sli)
    {
        for (fmy = 0; fmy < FMSIZE / 2 ; ++fmy)
        {
            for (fmx = 0; fmx < FMSIZE / 2 ; ++fmx)
            {
                outNeuIdx = sli * fmArea + fmy * 2 * FMSIZE + fmx * 2;
                max = devOutputNeuron[outNeuIdx];
                for (y = 0; y < 2; ++y)
                {
                    for (x = 0; x < 2; ++x)
                    {
                        ofmy = fmy * 2 + y;
                        ofmx = fmx * 2 + x;
                        outNeuIdx = sli * fmArea + ofmy * FMSIZE + ofmx;
                        tmpVal = devOutputNeuron[outNeuIdx];
                        if (tmpVal > max)
                        {
                            max = tmpVal;
                        }
                    }
                }
                outIdx = sli * outArea + fmy * FMSIZE / 2 + fmx;
                devOutput[outIdx] = max;
            }
        }
    }
}
/*** Implement your CUDA Kernel here ***/

int main()
{
  int convLayerCPUExecTime, convLayerGPUExecTime;
    init(); // Initialize the data on host memory

  timespec time_begin, time_end;

  clock_gettime(CLOCK_REALTIME, &time_begin);
    convLayerCPU();
  clock_gettime(CLOCK_REALTIME, &time_end);
  convLayerCPUExecTime = timespec_diff_us(time_begin, time_end);
  cout << "CPU time for executing a typical convolutional layer = " <<  convLayerCPUExecTime / 1000 << "ms" << endl;

    // declare device pointer
    short * devInputNeuron;
    short * devInputFilter;
    int * devOutputNeuron;
    int * devOutput;

    // compute the size for allocating memory on device
    const int inputNeuronSize = sizeof(short) * FMSIZE * FMSIZE * FMDEPTH;
    const int filtersSize = sizeof(short) * FILTNUM * FILTSIZE * FILTSIZE * FMDEPTH;
    const int outputNeuronSize = sizeof(int) * FMSIZE * FMSIZE * FMDEPTH;
    const int outputSize = sizeof(int) * FMSIZE / 2 * FMSIZE / 2 * FMDEPTH;

    // allocate memory on device
    hipMalloc(&devInputNeuron, inputNeuronSize);
    hipMalloc(&devInputFilter, filtersSize);
    hipMalloc(&devOutputNeuron, outputNeuronSize);
    hipMalloc(&devOutput, outputSize);

    // copy data from host to deivce
    hipMemcpy(devInputNeuron, inNeu, inputNeuronSize, hipMemcpyHostToDevice);
    hipMemcpy(devInputFilter, filt, filtersSize, hipMemcpyHostToDevice);

    /*** Lunch your CUDA Kernel here ***/
  clock_gettime(CLOCK_REALTIME, &time_begin);
    convLayerGPU<<<512, 1>>>(devInputNeuron, devInputFilter, devOutputNeuron, devOutput); // Lunch the kernel
    hipDeviceSynchronize(); // Do synchronization before clock_gettime()
  clock_gettime(CLOCK_REALTIME, &time_end);
    /*** Lunch your CUDA Kernel here ***/

    // copy data from device back to host
    hipMemcpy(outGPU, devOutput, outputSize, hipMemcpyDeviceToHost);

    // free the allocated memory on device
    hipFree(&devInputNeuron);
    hipFree(&devOutputNeuron);
    hipFree(&devOutput);
    hipFree(&devInputFilter);

  convLayerGPUExecTime = timespec_diff_us(time_begin, time_end);
  cout << "GPU time for executing a typical convolutional layer = " << convLayerGPUExecTime / 1000 << "ms" << endl;

    if (checker())
    {
        cout << "Congratulations! You pass the check." << endl;
        cout << "Speedup: " << (float)convLayerCPUExecTime / convLayerGPUExecTime << endl;
    }
    else
    {
        cout << "Sorry! Your result is wrong." << endl;
    }

    ending();

    return 0;
}
