#include "hip/hip_runtime.h"
// This program executes a typical convolutional layer in regular CNNs
#include <iostream>
#include "cnnConvLayer.h"

using namespace std;

static const int FMAREA(FMSIZE * FMSIZE);
static const int FILTAREA(FILTSIZE * FILTSIZE);
static const int FILTVOL(FMDEPTH * FILTAREA);

// This is the CPU version, please don't modify it
void convLayerCPU()
{
  const int filtVol = FMDEPTH * FILTSIZE * FILTSIZE;
  const int filtArea = FILTSIZE * FILTSIZE;
  const int fmArea = FMSIZE * FMSIZE;
  const int outArea = FMSIZE / 2 * FMSIZE / 2;

  int fn(0), fmy(0), fmx(0), sum(0), sli(0), y(0), x(0), ifmy(0), ifmx(0), \
    filtIdx(0), inNeuIdx(0), outNeuIdx(0), max(0), tmpVal(0), ofmy(0), ofmx(0), outIdx(0);

    // Convolution
  for (fn = 0; fn < FILTNUM; ++fn)
  {
    for (fmy = 0; fmy < FMSIZE; fmy += STRIDE)
    {
      for (fmx = 0; fmx < FMSIZE; fmx += STRIDE)
      {
        sum = 0;
        for (sli = 0; sli < FMDEPTH; ++sli)
        {
          for (y = 0; y < FILTSIZE; ++y)
          {
            for (x = 0; x < FILTSIZE; ++x)
            {
              ifmy = fmy - FILTSIZE / 2 + y;
              ifmx = fmx - FILTSIZE / 2 + x;
              filtIdx = fn * filtVol + sli * filtArea + y * FILTSIZE + x;
              inNeuIdx = sli * fmArea + ifmy * FMSIZE + ifmx;
              if (ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)
              {
                sum += filt[filtIdx] * inNeu[inNeuIdx];
              }
            }
          }
        }

        // Activation - ReLU
        outNeuIdx = fn * fmArea + fmy * FMSIZE + fmx;
        if (sum <= 0)
        {
          outNeu[outNeuIdx] = 0;
        }
        else
        {
          outNeu[outNeuIdx] = sum;
        }
      }
    }
  }

  // Max Pooling with Window Size 2x2
  for (sli = 0; sli < FILTNUM; ++sli)
  {
    for (fmy = 0; fmy < FMSIZE / 2 ; ++fmy)
    {
      for (fmx = 0; fmx < FMSIZE / 2 ; ++fmx)
      {
        outNeuIdx = sli * fmArea + fmy * 2 * FMSIZE + fmx * 2;
        max = outNeu[outNeuIdx];
        for (y = 0; y < 2; ++y)
        {
          for (x = 0; x < 2; ++x)
          {
            ofmy = fmy * 2 + y;
            ofmx = fmx * 2 + x;
            outNeuIdx = sli * fmArea + ofmy * FMSIZE + ofmx;
            tmpVal = outNeu[outNeuIdx];
            if (tmpVal > max)
            {
              max = tmpVal;
            }
          }
        }
        outIdx = sli * outArea + fmy * FMSIZE / 2 + fmx;
        outCPU[outIdx] = max;
      }
    }
  }
}

/*** Implement your CUDA Kernel here ***/
__global__
void convLayerGPU(short * inNeu, short * filt, int * out)
{
  __shared__ short filter[FILTAREA];
  __shared__ int outcome[1024], current_need_inNeu[1024];

  const int threadId(threadIdx.x), x(threadId % 32), y(threadId / 32), \
    outx(threadId % 16), outy(threadId / 16), outy_double(outy * 2), outx_double(outx * 2);
  int inx(0), iny(0), max(0);

  outcome[threadId] = 0;

  for (int slice(0); slice < 512; ++slice)
  {
    if (threadId < 9)
    {
      filter[threadId] = filt[FILTVOL * blockIdx.x + FILTAREA * slice + threadId];
    }

    current_need_inNeu[threadId] = inNeu[slice * FMAREA + threadId];

    __syncthreads();

    for (int ky(0); ky < FILTSIZE; ++ky)
    {
      for (int kx(0); kx < FILTSIZE; ++kx)
      {
        inx = x - FILTSIZE / 2 + kx;
        iny = y - FILTSIZE / 2 + ky;
        if (inx >= 0 && inx < FMSIZE && iny >= 0 && iny < FMSIZE)
        {
          outcome[threadId] += current_need_inNeu[iny * FMSIZE + inx] \
            * filter[ky * FILTSIZE + kx];
        }
      }
    }
    __syncthreads();
  }

    // Max Pooling with Window Size 2x2
  if (threadId < 256)
  {
    for (int i(0); i < 2; ++i)
    {
      for (int j(0); j < 2; ++j)
      {
        if (max < outcome[(outy_double + i) * FMSIZE + (outx_double + j)])
        {
          max = outcome[(outy_double + i) * FMSIZE + (outx_double + j)];
        }
      }
    }
    out[blockIdx.x * 256 + outy * 16 + outx] = max;
  }
}
/*** Implement your CUDA Kernel here ***/

int main()
{
  int convLayerCPUExecTime, convLayerGPUExecTime;
  init(); // Initialize the data on host memory

  timespec time_begin, time_end;

  clock_gettime(CLOCK_REALTIME, &time_begin);
  convLayerCPU();
  clock_gettime(CLOCK_REALTIME, &time_end);
  convLayerCPUExecTime = timespec_diff_us(time_begin, time_end);
  cout << "CPU time for executing a typical convolutional layer = " <<  convLayerCPUExecTime / 1000 << "ms" << endl;

  // declare device pointer
  short * devInputNeuron;
  short * devInputFilter;
  int * devOutputNeuron;
  int * devOutput;

  // compute the size for allocating memory on device
  const int inputNeuronSize = sizeof(short) * FMSIZE * FMSIZE * FMDEPTH;
  const int filtersSize = sizeof(short) * FILTNUM * FILTSIZE * FILTSIZE * FMDEPTH;
  const int outputSize = sizeof(int) * FMSIZE / 2 * FMSIZE / 2 * FMDEPTH;

  // allocate memory on device
  hipMalloc(&devInputNeuron, inputNeuronSize);
  hipMalloc(&devInputFilter, filtersSize);
  hipMalloc(&devOutput, outputSize);

  // copy data from host to deivce
  hipMemcpy(devInputNeuron, inNeu, inputNeuronSize, hipMemcpyHostToDevice);
  hipMemcpy(devInputFilter, filt, filtersSize, hipMemcpyHostToDevice);

  /*** Lunch your CUDA Kernel here ***/
  clock_gettime(CLOCK_REALTIME, &time_begin);
  convLayerGPU<<<512, 1024>>>(devInputNeuron, devInputFilter, devOutput); // Lunch the kernel
  hipDeviceSynchronize(); // Do synchronization before clock_gettime()
  clock_gettime(CLOCK_REALTIME, &time_end);
  /*** Lunch your CUDA Kernel here ***/

  // copy data from device back to host
  hipMemcpy(outGPU, devOutput, outputSize, hipMemcpyDeviceToHost);

  // free the allocated memory on device
  hipFree(&devInputNeuron);
  hipFree(&devOutputNeuron);
  hipFree(&devOutput);
  hipFree(&devInputFilter);

  convLayerGPUExecTime = timespec_diff_us(time_begin, time_end);
  cout << "GPU time for executing a typical convolutional layer = " << convLayerGPUExecTime / 1000 << "ms" << endl;

  if (checker())
  {
    cout << "Congratulations! You pass the check." << endl;
    cout << "Speedup: " << (float)convLayerCPUExecTime / convLayerGPUExecTime << endl;
  }
  else
  {
    cout << "Sorry! Your result is wrong." << endl;
  }

  ending();

  return 0;
}
